#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"
#include <fstream> 
#include "bitmap.h"
const int Tile_width = 16;

__constant__ double filter_d1[25];//Constant memory variable

texture<unsigned char,2,hipReadModeElementType> texIn; // Input to texture memory

__global__ void BlurGPUKernel(unsigned char* imaged, unsigned char* outputImaged,int width,int height,double* filter){

	int row = blockIdx.y * Tile_width + threadIdx.y;
	int col = blockIdx.x * Tile_width + threadIdx.x;

	if(row < height && col < width){
			//Perform Image convolution 
		double accum = 0;
			for(int fw = 0 ; fw < 5; fw++)
				for(int fh = 0; fh < 5; fh++)
				{
					int ix = ( col - 2 + fw + width)% width;
					int iy = ( row - 2 + fh + height)%height;
					accum = accum + (imaged[iy * width + ix] * filter[fw*5 + fh]);
				}
			accum /= 13;
			unsigned char temp = accum;
			outputImaged[row * width + col] = temp;
	}
}

__global__ void BlurGPUKernel1(unsigned char* imaged, unsigned char* outputImaged,int width,int height){

	int row = blockIdx.y * Tile_width + threadIdx.y;
	int col = blockIdx.x * Tile_width + threadIdx.x;

	if(row < height && col < width){
			//Perform Image convolution 
		double accum = 0;
			for(int fw = 0 ; fw < 5; fw++)
				for(int fh = 0; fh < 5; fh++)
				{
					int ix = ( col - 2 + fw + width)% width;
					int iy = ( row - 2 + fh + height)%height;
					accum = accum + (imaged[iy * width + ix] * filter_d1[fw*5 + fh]);
				}
			accum /= 13;
			unsigned char temp = accum;
			outputImaged[row * width + col] = temp;
	}
}

__global__ void BlurGPUKernel3( unsigned char* outputImaged,int width,int height,double* filter){

	int row = blockIdx.y * Tile_width + threadIdx.y;
	int col = blockIdx.x * Tile_width + threadIdx.x;

	if(row < height && col < width){
			//Perform Image convolution 
		double accum = 0;
			for(int fw = 0 ; fw < 5; fw++)
				for(int fh = 0; fh < 5; fh++)
				{
					int ix = ( col - 2 + fw + width)% width;
					int iy = ( row - 2 + fh + height)%height;
					accum = accum + (tex2D(texIn,ix,iy) * filter[fw*5 + fh]);
				}
			accum /= 13;
			unsigned char temp = accum;
			outputImaged[row * width + col] = temp;
	}
}

bool  BlurGPU( Bitmap* image, Bitmap* outputImage, int choice ){
	
	// Error return value
	hipError_t status;

	//size of matrix
	int size = image->Height() * image->Width();
	int bytes = size * sizeof(char);
	int bytes1 = 25 * sizeof(double);

	//Device pointers
	unsigned char *image_d;
	unsigned char *Outputimage_d;
	double *filter_d;

	// The width and height of the input image
	int width = image->Width();
	int height = image->Height();

	hipArray* carray;// Input data array
	hipChannelFormatDesc channel; //create channel to describe data type
	channel = hipCreateChannelDesc<unsigned char>(); 

	// blur filter
	double filter[5][5] =
	{
		 0, 0, 1, 0, 0,
		 0, 1, 1, 1, 0,
		 1, 1, 1, 1, 1,
		 0, 1, 1, 1, 0,
		 0, 0, 1, 0, 0,
	};


	

	//Allocation of device variables

	hipMallocArray(&carray,&channel,width,height);
	hipMalloc((void**)&image_d,bytes);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	     
	
	}
	hipMalloc((void**)&Outputimage_d,bytes);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	     
	
	}

	hipMalloc((void**)&filter_d,bytes1);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	     
	
	}

		// Copies the required input from Host to device
	hipMemcpy(image_d, image->image, bytes, hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Copy of image failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	}

	hipMemcpy(filter_d,filter, bytes1, hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Copy of filter failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	}

	
	hipMemcpyToSymbol(HIP_SYMBOL(filter_d1), filter, bytes1, 0, hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Copy of constant filter failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			hipFree(filter_d);
			return false;
	}

	hipMemcpyToArray(carray,0,0,image->image,bytes,hipMemcpyHostToDevice); // Copy required data to Array
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Copy of image failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	}

	//Set Texture address mode property
	texIn.addressMode[0]=hipAddressModeWrap;
	texIn.addressMode[1]=hipAddressModeClamp;
	hipBindTextureToArray(texIn,carray);

	dim3 dimBlock(Tile_width,Tile_width); //  Dimension of block
	dim3 dimGrid((int)ceil((float)width / (float)Tile_width), (int)ceil((float)height / (float)Tile_width)); // Dynamic allocation for dimension of grid
	if(choice == 1)
	BlurGPUKernel<<<dimGrid, dimBlock>>>(image_d,Outputimage_d,width,height,filter_d); // Kernel call
	else if(choice == 2)
	BlurGPUKernel1<<<dimGrid, dimBlock>>>(image_d,Outputimage_d,width,height);
	else if(choice == 3)
	BlurGPUKernel3<<<dimGrid, dimBlock>>>(Outputimage_d,width,height,filter_d);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Kernel failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	     
	
	}

	hipDeviceSynchronize(); // Cuda synchronize

	hipUnbindTexture (&texIn); // Unbind texture memory
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Unbind failed: " << hipGetErrorString(status) <<	std::endl;
			
			return false;     
		}

	hipMemcpy(outputImage->image, Outputimage_d, bytes, hipMemcpyDeviceToHost); // Copies the output form host to device.
	//Freeing allocated memory.

	hipFree(image_d);
	hipFree(Outputimage_d);
	hipFree(filter_d);

	return true; 
	
	}



