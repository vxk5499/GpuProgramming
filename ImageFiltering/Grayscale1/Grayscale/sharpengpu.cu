#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"
#include <fstream> 
#include <>
#include <hip/hip_texture_types.h>
#include "bitmap.h"
const int Tile_width = 16;

__constant__ double filter_d1[9]; //Constant memory variable

texture<unsigned char,2,hipReadModeElementType> texIn; // Input to texture memory

__global__ void SharpenGPUKernel(unsigned char* imaged, unsigned char* outputImaged,int width,int height,double* filter){

	int row = blockIdx.y * Tile_width + threadIdx.y;
	int col = blockIdx.x * Tile_width + threadIdx.x;

	if(row < height && col < width){
			//Perform Image convolution 
		double accum = 0;
			for(int fw = 0 ; fw < 3; fw++)
				for(int fh = 0; fh < 3; fh++)
				{
					int ix = ( col - 1 + fw + width)% width;
					int iy = ( row - 1 + fh + height)%height;
					accum = accum + (imaged[iy * width + ix] * filter[fw*3 + fh]);
				}
			unsigned char temp = accum;
			outputImaged[row * width + col] = temp;
	}
}

__global__ void SharpenGPUKernel1(unsigned char* imaged, unsigned char* outputImaged,int width,int height){

	int row = blockIdx.y * Tile_width + threadIdx.y;
	int col = blockIdx.x * Tile_width + threadIdx.x;

	if(row < height && col < width){
			//Perform Image convolution 
		double accum = 0;
			for(int fw = 0 ; fw < 3; fw++)
				for(int fh = 0; fh < 3; fh++)
				{
					int ix = ( col - 1 + fw + width)% width;
					int iy = ( row - 1 + fh + height)%height;
					accum = accum + (imaged[iy * width + ix] * filter_d1[fw*3 + fh]);
				}
			unsigned char temp = accum;
			outputImaged[row * width + col] = temp;
	}
}

__global__ void SharpenGPUKernel2( unsigned char* outputImaged,int width,int height,double* filter){

	int row = blockIdx.y * Tile_width + threadIdx.y;
	int col = blockIdx.x * Tile_width + threadIdx.x;

	if(row < height && col < width){
			//Perform Image convolution 
		double accum = 0;
			for(int fw = 0 ; fw < 3; fw++)
				for(int fh = 0; fh < 3; fh++)
				{
					int ix = ( col - 1 + fw + width)% width;
					int iy = ( row - 1 + fh + height)%height;
					accum = accum + (tex2D(texIn,ix,iy) * filter[fw*3 + fh]);
				}

			unsigned char temp = accum;
			outputImaged[row * width + col] = temp;
	}
}

bool  SharpenGPU( Bitmap* image, Bitmap* outputImage, int choice){
	
	// Error return value
	hipError_t status;

	

	hipArray* carray;// Input data array
	hipChannelFormatDesc channel; //create channel to describe data type
	channel = hipCreateChannelDesc<unsigned char>(); 
	
	//size of matrix
	int size = image->Height() * image->Width();
	int bytes = size * sizeof(unsigned char);
	int bytes1 = 9 * sizeof(double);

	//Device pointers
	unsigned char *image_d;
	unsigned char *Outputimage_d;
	double *filter_d;

	// The width and height of the input image
	int width = image->Width();
	int height = image->Height();

	hipMallocArray(&carray,&channel,width,height);

	// Sharpen filter
	double filter[9] =
	{
		0, -1, 0,
	    -1, 5,-1,
        0, -1, 0
	};
	

	//Allocation of device variables
	hipMalloc((void**)&image_d,bytes);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	 	}

	hipMalloc((void**)&Outputimage_d,bytes);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	     
	
	}

	hipMalloc((void**)&filter_d,bytes1);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			return false;
	     
	
	}

	hipMemcpyToArray(carray,0,0,image->image,bytes,hipMemcpyHostToDevice); // Copy required data to Array

	//Set Texture address mode property
	texIn.addressMode[0]=hipAddressModeWrap;
	texIn.addressMode[1]=hipAddressModeClamp;
	hipBindTextureToArray(texIn,carray);

	// Copies the required input from Host to device
	hipMemcpy(image_d, image->image, bytes, hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Copy of image failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			hipFree(filter_d);
			return false;
	}

	hipMemcpy(filter_d,filter, bytes1, hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Copy of filter failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			hipFree(filter_d);
			return false;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(filter_d1), filter, bytes1, 0, hipMemcpyHostToDevice);
	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Copy of constant filter failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			hipFree(filter_d);
			return false;
	}

	dim3 dimBlock(Tile_width,Tile_width); //  Dimension of block
	dim3 dimGrid((int)ceil((float)width / (float)Tile_width), (int)ceil((float)height / (float)Tile_width)); // Dynamic allocation for dimension of grid
	if (choice == 1)
	SharpenGPUKernel<<<dimGrid, dimBlock>>>(image_d,Outputimage_d,width,height,filter_d); // Kernel call
	else if(choice ==2)
	SharpenGPUKernel1<<<dimGrid, dimBlock>>>(image_d,Outputimage_d,width,height);
	else if(choice == 3)
	SharpenGPUKernel2<<<dimGrid, dimBlock>>>(Outputimage_d,width,height,filter_d);

	status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Kernel failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(image_d);
			hipFree(Outputimage_d);
			hipFree(filter_d);
			return false;
	     
	
	}

	hipDeviceSynchronize(); // Cuda synchronize

	hipUnbindTexture (&texIn); // Unbind texture memory
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Unbind failed: " << hipGetErrorString(status) <<	std::endl;
			
			return false;     
		}

	hipMemcpy(outputImage->image, Outputimage_d, bytes, hipMemcpyDeviceToHost); // Copies the output form host to device.
	//Freeing allocated memory.

	hipFree(image_d);
	hipFree(Outputimage_d);
	hipFree(filter_d);

	return true; 
	
	}



