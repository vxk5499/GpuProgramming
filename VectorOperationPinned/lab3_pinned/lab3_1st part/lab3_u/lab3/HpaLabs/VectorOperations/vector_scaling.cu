#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"

// GPU kernel to perform Vector Addition

__global__ void vector_scalingKernel(float* ad, float* cd, float scaleFactor, int size)
{
	// Retrive thread id within the block
	int th_id = threadIdx.x + blockIdx.x * blockDim.x;

	
		// Perform vector addition
	while(th_id<size) {
		cd[th_id] = ad[th_id] * scaleFactor;
		th_id= blockDim.x * gridDim.x;
	}
	
}

bool scaleVectorGPU( float* a, float* c, float scaleFactor, int size )

{
	// Error return value
	hipError_t status;
	// Number of bytes in a vector
	int bytes = size * sizeof(float);
	// Pointer to the device arrays
		float *ad, *cd;

	// Device pointer to pinned meory
	hipHostGetDevicePointer( (void**)&ad, a, 0 );
	
	hipHostGetDevicePointer( (void**)&cd, c, 0 );
	
	
	dim3 dimBlock(1024); //  is contained in a block
	dim3 dimGrid((size+1023)/1024);
	// Launch the kernel on a size-by-size block of threads
	vector_scalingKernel<<<dimGrid, dimBlock>>>(ad, cd, scaleFactor, size);
	hipDeviceSynchronize();// Sync threads
	 // Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
	std::cout << "Kernel failed: " << hipGetErrorString(status) <<	std::endl;
	
	return false;
	}
	
	// Success
	
	return true;
	

}

