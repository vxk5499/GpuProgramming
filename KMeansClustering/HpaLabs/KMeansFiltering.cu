#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "KMeans.h"
#include <fstream> 

__constant__ Vector2 clusters_d[3];

// Kernel to perform the assignment of data points to the clusters
__global__ void KMeansKernel ( Datapoint* data_d, long n, int k, int* flag )
{					
		
			int pos = 0;
			int th_id = blockIdx.x *blockDim.x + threadIdx.x;
			if(th_id<n){
			int ip = data_d[th_id].cluster;	 // Initial cluster for data point
			float dist1 = clusters_d[0].distSq(data_d[th_id].p); // Distance of the point from the center point of the cluster 1
			float dist2 = clusters_d[1].distSq(data_d[th_id].p); // Distance of the point from the center point of the cluster 2
			float dist3 = clusters_d[2].distSq(data_d[th_id].p); // Distance of the point from the center point of the cluster 3
			
			// Finding the smallest distance
			if(dist1<=dist2 && dist1<=dist3)
				pos = 0;
			
			if(dist2<=dist1 && dist2<=dist3)
				pos = 1;

			if(dist3<=dist2 && dist3<=dist1)
				pos = 2;

		
			// Assignment of the datapoint to the nearest cluster
			switch (pos){
			case 0: data_d[th_id].cluster = 0;
					break;
			case 1: data_d[th_id].cluster = 1;
					break;
			case 2: data_d[th_id].cluster = 2;
					break; 

			}
		

			int fp = data_d[th_id].cluster; // Final cluster for data point
			//Checking whether the data point has chnaged the cluster
			if( ip != fp){
				*flag = 0;}
			}
}
bool KMeansGPU( Datapoint* data, long n, Vector2* clusters, int k )
{
	hipError_t status; // Check for Error in Cuda

	int size = n * sizeof(Datapoint);
	int size1 = 3 * sizeof(Vector2);
	int size2 = sizeof(int);

	int *flag_d;

	Datapoint* data_d;
	//Allocation of device data
	hipMalloc((void**) &data_d, size);
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			
			hipFree(data_d);
			return false;     
		}
	//Allocation of device data
	hipMalloc((void**) &flag_d, size2);

	// Copying of data points to device memory from host
	hipMemcpy(data_d, data, size, hipMemcpyHostToDevice);
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc1 failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(clusters_d);
			hipFree(data_d);
			return false;     
		}
			hipMemcpyToSymbol(HIP_SYMBOL(clusters_d), clusters, size1, 0, hipMemcpyHostToDevice);
				status = hipGetLastError();


	dim3 dimBlock(768);
	dim3 dimGrid((int)ceil((float)n/(float)768));
	int flag = 0;
	while(flag == 0)
	{

		//float pin[3]={0,0,0};
		flag = 1;

		// Copying of data points to device memory from host
		hipMemcpy(data_d, data, size, hipMemcpyHostToDevice);
		if (status != hipSuccess) {
			std::cout << "Memcpy failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(data_d);
			return false;     
		}

		// Copying of flag to device memory from host
		hipMemcpy(flag_d, &flag, size2, hipMemcpyHostToDevice);
				status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Memcpy1 failed: " << hipGetErrorString(status) <<	std::endl;
			
			hipFree(data_d);
			return false;     
		}

		//Calling of Kernel
		KMeansKernel<<<dimGrid, dimBlock>>>(data_d, n, k,flag_d);
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Kernel failed: " << hipGetErrorString(status) <<	std::endl;
		
			hipFree(data_d);
			return false;     
		}

		hipDeviceSynchronize();// Thread synchronize

		//Copy of data points to host from device
		hipMemcpy(data, data_d, size, hipMemcpyDeviceToHost);
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Memcpy2 failed: " << hipGetErrorString(status) <<	std::endl;
			
			hipFree(data_d);
			return false;     
		}

		//Copy of flag to host from device
		hipMemcpy(&flag, flag_d, size2, hipMemcpyDeviceToHost);
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Memcpy3 failed: " << hipGetErrorString(status) <<	std::endl;
			
			hipFree(data_d);
			return false;     
		}
		
		int count; // To keep track of number of data points in cluster

		//Finding the centre in cluster
		for(int j = 0; j<k ; j++)
		{
			count = 0;
			float X = 0, Y = 0; // Variable to store the sum of values in x and y direction
			for(long i = 0; i < n; i++)
			{
				if(data[i].cluster == j)
				{
					X += data[i].p.x;
					Y += data[i].p.y;
					count += 1;
				}
			}
		clusters[j].x = X / count; // Finding cluster center in x direction
		clusters[j].y = Y / count; // Finding cluster center in y direction
		}

		// Copying of updated cluster values to constant memory
		hipMemcpyToSymbol(HIP_SYMBOL(clusters_d), clusters, size1, 0, hipMemcpyHostToDevice);
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Memcpy4 failed: " << hipGetErrorString(status) <<	std::endl;
			
			hipFree(data_d);
			return false;     
		}
		
	}

	hipFree(data_d);
	hipFree(flag_d);
	return true;

}