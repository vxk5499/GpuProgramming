#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"

// GPU kernel to perform Vector Addition

__global__ void vector_scalingKernel(float* ad, float* cd, float scaleFactor, int size)
{
	// Retrive thread id within the block
	int th_id = threadIdx.x + blockIdx.x * blockDim.x;

	
		// Perform vector addition
	while(th_id<size) {
		cd[th_id] = ad[th_id] * scaleFactor;
		th_id= blockDim.x * gridDim.x;
	}
	
}

bool scaleVectorGPU( float* a, float* c, float scaleFactor, int size )

{
	// Error return value
	hipError_t status;
	// Number of bytes in a vector
	int bytes = size * sizeof(float);
	// Pointer to the device arrays
	float *ad, *cd;
	// Allocate memory on the device to store each vector
	hipMalloc((void**) &ad, bytes);
	hipMalloc((void**) &cd, bytes);
	// Copy the host input data to the device
	hipMemcpy(ad, a, bytes, hipMemcpyHostToDevice);
	// Specify the size of the grid and the size of the block
	dim3 dimBlock(1024); //  is contained in a block
	dim3 dimGrid((size+1023)/1024); // Only using a single grid element
	// Launch the kernel on a size-by-size block of threads
	 vector_scalingKernel<<<dimGrid, dimBlock>>>(ad, cd, scaleFactor, size);
	 // Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
	std::cout << "Kernel failed: " << hipGetErrorString(status) <<	std::endl;
	hipFree(ad);
	hipFree(cd);
	return false;
	}
	// Retrieve the result vector
	hipMemcpy(c, cd, bytes, hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(ad);
	hipFree(cd);
	// Success
	
	return true;
	

}

