#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"

// GPU kernel to perform Vector Addition

__global__ void vector_subractionKernel(float* ad, float* bd, float* cd, int size)
{
	// Retrive thread id within the block
	int th_id = threadIdx.x + blockIdx.x * blockDim.x;

	
		// Perform vector subraction
	while(th_id<size)	{
		cd[th_id] = ad[th_id] - bd[th_id];
		th_id= blockDim.x * gridDim.x;
	}
}

bool subtractVectorGPU( float* a, float* b, float* c, int size )
{
	// Error return value
	hipError_t status;
	// Number of bytes in a vector
	int bytes = size * sizeof(float);
	// Pointer to the device arrays
	float *ad, *bd, *cd;
	// Allocate memory on the device to store each vector
	hipMalloc((void**) &ad, bytes);
	hipMalloc((void**) &bd, bytes);
	hipMalloc((void**) &cd, bytes);
	// Copy the host input data to the device
	hipMemcpy(ad, a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, bytes, hipMemcpyHostToDevice);
	// Specify the size of the grid and the size of the block
	dim3 dimBlock(1024); //  is contained in a block
	dim3 dimGrid((size+1023)/1024); // Only using a single grid element
	// Launch the kernel on a size-by-size block of threads
	 vector_subractionKernel<<<dimGrid, dimBlock>>>(ad, bd, cd, size);
	 // Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
	std::cout << "Kernel failed: " << hipGetErrorString(status) <<	std::endl;
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	return false;
	}
	// Retrieve the result vector
	hipMemcpy(c, cd, bytes, hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	// Success
	
	return true;
	

}

