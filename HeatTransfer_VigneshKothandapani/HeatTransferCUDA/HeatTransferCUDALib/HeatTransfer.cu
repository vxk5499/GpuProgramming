#include "hip/hip_runtime.h"
/************************************************************************/
// The purpose of this file is to provide a GPU implementation of the 
// heat transfer simulation using MATLAB.
//
// Author: Jason Lowden
// Date: October 20, 2013
//
// File: KMeans.h
/************************************************************************/
#include <hip/hip_runtime_api.h>
#include <>
#include <>
#include <hip/hip_texture_types.h>
#include <iostream>

#include "HeatTransfer.h"

texture<float,2> texIn; // Input to texture memory

__global__ void UpdateHeatMapKernel(float * texOut,int size, float heatSpeed)
{
	
	int col = threadIdx.x + blockIdx.x * blockDim.x; // Calculates the current column
	int row = threadIdx.y + blockIdx.y * blockDim.y; // Calculate the current row
	int offset = col + row * size; // indicates the cureent operating element
	if(col > 0 && col < size-1 && row < size-1 && row > 0){
		
	
	float top = tex2D(texIn, col, row-1); // element on top of current element
	float left = tex2D(texIn, col-1, row);  // element on left of current element
	float right = tex2D(texIn, col+1, row); // element on right of current element
	float bottom = tex2D(texIn, col, row+1);// element on bottom of current element
	float current = tex2D(texIn, col, row); // Current element
	float temp =  heatSpeed * ( top + bottom + right + left - (4 * current)); // heat transfeered from other elements
	
	texOut[offset] = current + temp; // New heat

		
	}
}

// Calculates the updated heat map for a given size based on number of iterations
bool UpdateHeatMap(float* dataIn, float* dataOut, int size, float heatSpeed, int numIterations)
{
	hipError_t status; // to check success of cuda calls

	int bytes = size * size * sizeof(float); // size of input data

	hipArray_t dataIn_d; // Input data array

	float* texOut; //Output from texture memory

	//Allocation of device data
	hipMalloc((void**)&texOut, bytes);
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Alloc failed: " << hipGetErrorString(status) <<	std::endl;
			
			hipFree(texOut);
			return false;     
		}

	//Copying data to device memory
	hipMemcpy(texOut, dataIn, bytes, hipMemcpyHostToDevice);
	status = hipGetLastError();
	if (status != hipSuccess) {
			std::cout << "Memcopy failed failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	status = hipGetLastError();
	if (status != hipSuccess) {
			std::cout << "Desc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}
	unsigned int flags=0;
	// Allocate array in device
	hipMallocArray(&dataIn_d, &desc,size, size);
	status = hipGetLastError();
	if (status != hipSuccess) {
			std::cout << "Array alloc failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}
	int size1= size * size *sizeof(float);
	//Copy data into array
	hipMemcpyToArray (dataIn_d, 0, 0, dataIn, size1, hipMemcpyHostToDevice);
	status = hipGetLastError();
	if (status != hipSuccess) {
			std::cout << "memcpy to array failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}

	//Bind array to texture
	hipBindTextureToArray (&texIn, dataIn_d, &desc);
	status = hipGetLastError();
	if (status != hipSuccess) {
			std::cout << "Cuda Binding failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}

	dim3 dimBlock(16,16); //  Dimension of block
	dim3 dimGrid((int)ceil((float)size / (float)16), (int)ceil((float)size / (float)16)); // Dynamic allocation for dimension of grid


	for(int i = 0; i < numIterations; i++)
	{
		UpdateHeatMapKernel<<<dimGrid, dimBlock>>>(texOut, size, heatSpeed); // Calls heat map Kernel
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Cuda kernal failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}
		hipDeviceSynchronize(); // Cuda Synchronisation 
		status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Sync failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}
		hipUnbindTexture (&texIn); // Unbind texture memory
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Unbind failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}
		hipMemcpyToArray (dataIn_d, 0, 0, texOut, size1, hipMemcpyDeviceToDevice); // Cuda memcpy to array within device
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "memcpy to array failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}
		hipBindTextureToArray (&texIn, dataIn_d, &desc); // Bind array to texture memory
			status = hipGetLastError();
		if (status != hipSuccess) {
			std::cout << "Bind failed: " << hipGetErrorString(status) <<	std::endl;
			hipFree(texOut);
			return false;     
		}

	}

	hipMemcpy(dataOut, texOut, bytes, hipMemcpyDeviceToHost); // Copy results to host
	hipUnbindTexture (&texIn); // Unbind texture memory
	hipFree(texOut); // Free cuda memory
	hipFreeArray(dataIn_d); // Free cuda memory
	





	return true;
}